#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

char *fileIN, *fileOUT;
unsigned char *image;
int width, height, pixelWidth; //meta info de la imagen

#ifndef SIZE
#define SIZE 1024
#endif

#ifndef PINNED
#define PINNED 0
#endif


__global__ void MaxPooling (int height, int width, int channels, int kernel_X, int kernel_Y, int *I, int *I_out)
{
  int x = (blockIdx.x * blockDim.x + threadIdx.x) * kernel_X;
  int y = (blockIdx.y * blockDim.y + threadIdx.y) * kernel_Y;

  int max_sum = 0;

  if (x < height and y < width)
  {
    for (int i = x; (i < height) and (i < i + kernel_X); ++i)
    {
      for (int j = y; (j < width) and (j < j + kernel_Y); ++j)
      {
        int sum_chanels = I[i*width + j + 0] + I[i*width + j + 1] + I[i*width + j + 2];
        if (sum_chanels > max_sum)
        {
          for (int c = 0; c < channels; ++c)
          {
            *I_out[(i/kernel_X)*width + j/kernel_Y + c] = I[i*width + j + c];
          }
        }
      }
    }
  }
}


int main(int argc, char** argv)
{
  // Ficheros de entrada, de salida, tamaño filtrado y num GPUs
  if (argc == 3) { fileIN = argv[1]; fileOUT = argv[2]; kernelSize = argv[3]; numgpu = argv[4]}
  else { printf("Usage: ./exe fileIN fileOUT kernelSize numGPUs\n"); exit(0); }


  printf("Reading image...\n");
  image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);
  if (!image) {
    fprintf(stderr, "Couldn't load image.\n");
     return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);

  int count;
  hipGetDeviceCount(&count);

  if (count < numgpu) { printf("No hay suficientes GPUs\n"); exit(0); }
  
  /*
  nThreads_X = width/kernelSize; // 320 / 2 = 160
  nThreads_Y = height/kernelSize; // 640 / 2 = 320
  nBlocks_X = width/nThreads; //
  nBlocks_Y = height/nThreads;
  */

  nThreads_X = (width%kernelSize == 0) ? width/kernelSize : width/kernelSize + 1;
  nThreads_Y = (height%kernelSize == 0) ? height/kernelSize : height/kernelSize + 1;
  nThreads = nThreads_X * nThreads_Y;
  nBlocks = nThreads/SIZE;

  dim3 dimGrid(nBlocks, nBlocks, 1);
  dim3 dimBlock(nThreads, nThreads, 1);

  hipEvent_t E0, E1, E2, E3;
  float TiempoTotal, TiempoKernel;

  //imagen original en el device
  int *d_image;
  //imagen modificada en el device
  int *d_image_out;

  //imagen modificada en el host
  int *I_out;

  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  // Obtener Memoria en el host de la imagen resultante
  I_out = (int*) malloc(width*height/kernelSize);
  //I_out = (int*) malloc(width*height/kernelSize*numgpu);

  // Obtiene Memoria [pinned] en el host
  //hipHostMalloc((float**)&I_out, numBytes);
  //hipHostMalloc((float**)&H_y, numBytes);   // Solo se usa para comprobar el resultado

 
  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
 
  // Obtener Memoria en el device de la imagen original y la resultante
  hipMalloc((int**)&d_image, width*height);
  hipMalloc((int**)&d_image_out, width*height/kernelSize);
  CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 

  // Copiar datos desde el host en el device 
  hipMemcpy(d_image, image, width*height, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);

  // Ejecutar el kernel 
  MaxPooling<<<nBlocks, nThreads>>>(height, width, 3, kernelSize, kernelSize, d_image, d_image_out);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);

  // Obtener el resultado desde el host 
  // Guardamos el resultado en I_out para poder comprobar el resultado
  hipMemcpy(I_out, d_image_out, width*height/kernelSize, hipMemcpyDeviceToHost); 
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

  // Liberar Memoria del device 
  hipFree(d_image); hipFree(d_image_out);

  hipDeviceSynchronize();

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal,  E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
 
  printf("nThreads: %d\n", nThreads);
  printf("nBlocks: %d\n", nBlocks);

  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

  printf("Filtrando\n");
	//SECUENCIAL BLANCO Y NEGRO:
	/*for(int i=0;i<width*height*3;i=i+3){
		image[i]=image[i];
		image[i+1]=image[i+1];
		image[i+2]=0;
	}*/



  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,image,0);

}

